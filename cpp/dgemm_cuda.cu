#include "hip/hip_runtime.h"
#include <hipblas.h>
#include "dgemm.h"

void catchCudaError(const char* file, int line) {
    hipError_t e = hipGetLastError();
    if (e != hipSuccess) {
        printf("Cuda-Error: %s %s %d\n", hipGetErrorString(e), file, line);
        hipDeviceReset();
        exit(0);
    }
}

__global__ void cuda_sgemm(float* matrix_a,
                           float* matrix_b,
                           float* matrix_c,
                           size_t M,
                           size_t K,
                           size_t N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float sum = 0;
    if (col < N && row < M) {
        for (int k = 0; k < K; k++) {
            sum +=
                matrix_a[INDEX(row, k, M, K)] * matrix_b[INDEX(k, col, K, N)];
        }
        matrix_c[INDEX(row, col, M, N)] = sum;
    }
}

__global__ void cuda_dgemm(double* matrix_a,
                           double* matrix_b,
                           double* matrix_c,
                           size_t M,
                           size_t K,
                           size_t N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    double sum = 0;
    if (col < N && row < M) {
        for (int k = 0; k < K; k++) {
            sum +=
                matrix_a[INDEX(row, k, M, K)] * matrix_b[INDEX(k, col, K, N)];
        }
        matrix_c[INDEX(row, col, M, N)] = sum;
    }
}

void dgemm::sgemm_cuda_loops(double* matrix_a,
                             double* matrix_b,
                             double* result,
                             int M,
                             int K,
                             int N,
                             int repeats) {
    size_t sizeA = M * K;
    size_t sizeB = K * N;
    size_t sizeC = M * N;

    float* a = (float*)malloc(sizeA * sizeof(float));
    for (size_t i = 0; i < sizeA; i++)
        a[i] = (float)matrix_a[i];

    float* b = (float*)malloc(sizeB * sizeof(float));
    for (size_t i = 0; i < sizeB; i++)
        b[i] = (float)matrix_b[i];

    float* c = (float*)malloc(sizeC * sizeof(float));

    // allocate array in gpu memory
    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, sizeA * sizeof(float));
    hipMalloc(&d_b, sizeB * sizeof(float));
    hipMalloc(&d_c, sizeC * sizeof(float));

    hipMemcpy(d_a, a, sizeA * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeB * sizeof(float), hipMemcpyHostToDevice);
    catchCudaError(__FILE__, __LINE__);

    size_t block_size = 16;
    size_t grid_rows = ceil((double)M / block_size);
    size_t grid_cols = ceil((double)N / block_size);
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(block_size, block_size);

    // run gpu kernel
    for (int r = 0; r < repeats; r++)
        cuda_sgemm<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, M, K, N);

    catchCudaError(__FILE__, __LINE__);

    // copy results back
    hipMemcpy(c, d_c, sizeC * sizeof(float), hipMemcpyDeviceToHost);

    // Free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();
    catchCudaError(__FILE__, __LINE__);

    for (size_t i = 0; i < sizeC; i++)
        result[i] = (double)c[i];
    free(a);
    free(b);
    free(c);
}

void dgemm::dgemm_cuda_loops(double* matrix_a,
                             double* matrix_b,
                             double* result,
                             int M,
                             int K,
                             int N,
                             int repeats) {
    size_t sizeA = M * K;
    size_t sizeB = K * N;
    size_t sizeC = M * N;

    // allocate array in gpu memory
    double *d_a, *d_b, *d_c;
    hipMalloc(&d_a, sizeA * sizeof(double));
    hipMalloc(&d_b, sizeB * sizeof(double));
    hipMalloc(&d_c, sizeC * sizeof(double));

    hipMemcpy(d_a, matrix_a, sizeA * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, matrix_b, sizeB * sizeof(double), hipMemcpyHostToDevice);
    catchCudaError(__FILE__, __LINE__);

    size_t block_size = 16;
    size_t grid_rows = ceil((double)M / block_size);
    size_t grid_cols = ceil((double)N / block_size);
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(block_size, block_size);

    // run gpu kernel
    for (int r = 0; r < repeats; r++)
        cuda_dgemm<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, M, K, N);

    catchCudaError(__FILE__, __LINE__);

    // copy results back
    hipMemcpy(result, d_c, sizeC * sizeof(double), hipMemcpyDeviceToHost);

    // Free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();
    catchCudaError(__FILE__, __LINE__);
}

void dgemm::sgemm_cuda_cublas(double* matrix_a,
                              double* matrix_b,
                              double* result,
                              int M,
                              int K,
                              int N,
                              int repeats) {
    size_t sizeA = M * K;
    size_t sizeB = K * N;
    size_t sizeC = M * N;

    float* a = (float*)malloc(sizeA * sizeof(float));
    for (size_t i = 0; i < sizeA; i++)
        a[i] = (float)matrix_a[i];

    float* b = (float*)malloc(sizeB * sizeof(float));
    for (size_t i = 0; i < sizeB; i++)
        b[i] = (float)matrix_b[i];

    float* c = (float*)malloc(sizeC * sizeof(float));

    // allocate array in gpu memory
    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, sizeA * sizeof(float));
    hipMalloc(&d_b, sizeB * sizeof(float));
    hipMalloc(&d_c, sizeC * sizeof(float));

    hipMemcpy(d_a, a, sizeA * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeB * sizeof(float), hipMemcpyHostToDevice);
    catchCudaError(__FILE__, __LINE__);

    size_t block_size = 16;
    size_t grid_rows = ceil((double)M / block_size);
    size_t grid_cols = ceil((double)N / block_size);
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(block_size, block_size);

    // Create a handle for CUBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // run gpu kernel
    for (int r = 0; r < repeats; r++) {
        float one = 1.0;
        float zero = 0.0;

// Do the actual multiplication
#ifdef COLUMN_MAJOR
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &one, d_a, M,
                    d_b, K, &zero, d_c, M);

#else
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &one, d_b, N,
                    d_a, K, &zero, d_c, N);
#endif
    }

    // Destroy the handle
    hipblasDestroy(handle);

    catchCudaError(__FILE__, __LINE__);

    // copy results back
    hipMemcpy(c, d_c, sizeC * sizeof(float), hipMemcpyDeviceToHost);

    // Free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();
    catchCudaError(__FILE__, __LINE__);

    for (size_t i = 0; i < sizeC; i++)
        result[i] = (double)c[i];
    free(a);
    free(b);
    free(c);
}

void dgemm::dgemm_cuda_cublas(double* matrix_a,
                              double* matrix_b,
                              double* result,
                              int M,
                              int K,
                              int N,
                              int repeats) {
    size_t sizeA = M * K;
    size_t sizeB = K * N;
    size_t sizeC = M * N;

    // allocate array in gpu memory
    double *d_a, *d_b, *d_c;
    hipMalloc(&d_a, sizeA * sizeof(double));
    hipMalloc(&d_b, sizeB * sizeof(double));
    hipMalloc(&d_c, sizeC * sizeof(double));

    hipMemcpy(d_a, matrix_a, sizeA * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, matrix_b, sizeB * sizeof(double), hipMemcpyHostToDevice);
    catchCudaError(__FILE__, __LINE__);

    size_t block_size = 16;
    size_t grid_rows = ceil((double)M / block_size);
    size_t grid_cols = ceil((double)N / block_size);
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(block_size, block_size);

    // Create a handle for CUBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // run gpu kernel
    for (int r = 0; r < repeats; r++) {
        double one = 1.0;
        double zero = 0.0;

// Do the actual multiplication
#ifdef COLUMN_MAJOR
        hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &one, d_a, M,
                    d_b, K, &zero, d_c, M);

#else
        hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &one, d_b, N,
                    d_a, K, &zero, d_c, N);
#endif
    }

    // Destroy the handle
    hipblasDestroy(handle);

    catchCudaError(__FILE__, __LINE__);

    // copy results back
    hipMemcpy(result, d_c, sizeC * sizeof(double), hipMemcpyDeviceToHost);

    // Free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();
    catchCudaError(__FILE__, __LINE__);
}
